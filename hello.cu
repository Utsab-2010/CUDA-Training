//hello.c


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu() {
    printf("Hello from GPU!\n");
}

int main() {
    hello_from_gpu<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
